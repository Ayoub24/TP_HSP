
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

void MatrixInit(float *M, int n, int p) {
    for(int i = 0; i < p; i++) {
        for(int j = 0; j < n; j++) {
            M[i*n+j] = float(rand())/(float(RAND_MAX)/2.0)-1.0;
        }
    }
}

void MatrixPrint(float *M, int n, int p){
    printf("[");
    for (int row=0; row<n; row++)
    {
        for(int col=0; col<p; col++)
        {
            if (row==n-1 & col==p-1){
                printf("%f]", M[row*p+col]);
            }else{
                printf("%f    ", M[row*p+col]);
            }
        }
        printf("\n");
    }
}

double MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    clock_t t;
    t = clock();
    for(int i = 0; i < p; i++) {
        for(int j = 0; j < n; j++) {
            Mout[i*n+j] = M1[i*n+j] + M2[i*n+j];
        }
    }
    t = clock() - t;
    printf("%ld\n\n", t);
    return double(t)/double(CLOCKS_PER_SEC);
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    clock_t t;
    t = clock();
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    while(tid < n*p) {
        Mout[tid] = M1[tid] + M2[tid];
        tid += blockDim.x;
    }
    t = clock() - t;
}

void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    int sum = 0;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            sum = 0;
            for(int k = 0; k < n; k++) {
                sum += M1[i*n+k]*M2[k*n+j];
            }
            Mout[i*n+j] = sum;
        }
    }
}

int main() {
    double CPUtime = 0;
    double *GPUtime = 0;
    int n = 5;
    int p = 5;
    float *M1;
    float *M2;
    float *M3;
    float *M4;
    M1 = (float *) malloc(sizeof(float)*p*n);
    M2 = (float *) malloc(sizeof(float)*p*n);
    M3 = (float *) malloc(sizeof(float)*p*n);
    M4 = (float *) malloc(sizeof(float)*p*n);
    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    CPUtime = MatrixAdd(M1, M2, M3, n, p);
    MatrixPrint(M1, n, p);
    printf("\n");
    MatrixPrint(M2, n, p);
    printf("\n");
    MatrixPrint(M3, n, p);
    printf("\n");

    float *cM1;
    float *cM2;
    float *cM3;
    if(hipMalloc((float**)&cM1, sizeof(float)*p*n) != hipSuccess) {
        fprintf(stderr, "hipMalloc failed.");
        exit(1);
    }
    if(hipMalloc((float**)&cM2, sizeof(float)*p*n) != hipSuccess) {
        fprintf(stderr, "hipMalloc failed.");
        exit(1);
    }
    if(hipMalloc((float**)&cM3, sizeof(float)*p*n) != hipSuccess) {
        fprintf(stderr, "hipMalloc failed.");
        exit(1);
    }
    if(hipMemcpy(cM1, M1, sizeof(float)*p*n, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed.");
        exit(1);
    }
    if(hipMemcpy(cM2, M2, sizeof(float)*p*n, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed.");
        exit(1);
    }

    cudaMatrixAdd<<<n,p>>>(cM1, cM2, cM3, n, p);
    if(hipMemcpy(M4, cM3, sizeof(float)*p*n, hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed.");
        exit(1);
    }

    MatrixPrint(M4, n, p);

    printf("\n\nCPU time : %f\n", CPUtime);

    free(M1);
    free(M2);
    free(M3);
    free(M4);
    hipFree(cM1);
    hipFree(cM2);
    hipFree(cM3);

    return 0;
}

